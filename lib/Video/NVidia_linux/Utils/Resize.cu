#include "hip/hip_runtime.h"
/*
* Copyright 2017-2018 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <hip/hip_runtime.h>
#include "NvCodecUtils.h"

// [BMH 2018-09-18]
// I added sRGB resizing for the Luma channel. If you don't do this, then the image gets darkened.
// HOWEVER, I don't know what the right thing to do is, for 16-bit data.
// This code naively assumes that the 16-bit data is also sRGB encoded, which I think is unlikely
// to be true.
#define sRGB_RESIZE

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}

template<class TOut>
__device__ static TOut LinearTosRGB(float v) {
    const float a = 0.055f;
    const float scale =  (1 << (sizeof(TOut) * 8)) - 1;
	v = v <= 0.0031308f ? 12.92f * v : (1.0f + a) * pow(v, (1.0f / 2.4f));
	return (TOut) Clamp(v * scale, 0.0f, scale);
}

template<typename YuvUnitx2>
static __global__ void Resize(hipTextureObject_t texY, hipTextureObject_t texUv,
        uint8_t *pDst, uint8_t *pDstUV, int nPitch, int nWidth, int nHeight,
        float fxScale, float fyScale)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x,
        iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= nWidth / 2 || iy >= nHeight / 2) {
        return;
    }

    int x = ix * 2;
    int y = iy * 2;
    typedef decltype(YuvUnitx2::x) YuvUnit;
    const int MAX = (1 << (sizeof(YuvUnit) * 8)) - 1;
    
#ifdef sRGB_RESIZE
    YuvUnit* dstY = (YuvUnit*) (pDst + y * nPitch + x * sizeof(YuvUnit));
    dstY[0] = LinearTosRGB<YuvUnit>(tex2D<float>(texY, x / fxScale, y / fyScale));
    dstY[1] = LinearTosRGB<YuvUnit>(tex2D<float>(texY, (x + 1) / fxScale, y / fyScale));
    
    y++;
    dstY += nPitch / sizeof(YuvUnit);
    dstY[0] = LinearTosRGB<YuvUnit>(tex2D<float>(texY, x / fxScale, y / fyScale));
    dstY[1] = LinearTosRGB<YuvUnit>(tex2D<float>(texY, (x + 1) / fxScale, y / fyScale));
#else
    YuvUnit* dstY = (YuvUnit*) (pDst + y * nPitch + x * sizeof(YuvUnit));
    dstY[0] = (YuvUnit) (tex2D<float>(texY, x / fxScale, y / fyScale) * MAX);
    dstY[1] = (YuvUnit) (tex2D<float>(texY, (x + 1) / fxScale, y / fyScale) * MAX);

    y++;
    dstY += nPitch / sizeof(YuvUnit);
    dstY[0] = (YuvUnit) (tex2D<float>(texY, x / fxScale, y / fyScale) * MAX);
    dstY[1] = (YuvUnit) (tex2D<float>(texY, (x + 1) / fxScale, y / fyScale) * MAX);
#endif

    float2 uv = tex2D<float2>(texUv, ix / fxScale, iy / fyScale);
    *(YuvUnitx2 *)(pDstUV + iy * nPitch + ix * sizeof(YuvUnitx2)) = YuvUnitx2{ (YuvUnit)(uv.x * MAX), (YuvUnit)(uv.y * MAX) };
}

template <typename YuvUnitx2>
static void Resize(unsigned char *dpDst, unsigned char* dpDstUV, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight) {
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dpSrc;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<decltype(YuvUnitx2::x)>();
    resDesc.res.pitch2D.width = nSrcWidth;
    resDesc.res.pitch2D.height = nSrcHeight;
    resDesc.res.pitch2D.pitchInBytes = nSrcPitch;

    hipTextureDesc texDesc = {};
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;
#ifdef sRGB_RESIZE
    texDesc.sRGB = 1;
#endif

    hipTextureObject_t texY = 0;
    ck(hipCreateTextureObject(&texY, &resDesc, &texDesc, NULL));

    resDesc.res.pitch2D.devPtr = dpSrc + nSrcPitch * nSrcHeight;
    resDesc.res.pitch2D.desc = hipCreateChannelDesc<YuvUnitx2>();
    resDesc.res.pitch2D.width = nSrcWidth / 2;
    resDesc.res.pitch2D.height = nSrcHeight / 2;
    texDesc.sRGB = 0;

    hipTextureObject_t texUv = 0;
    ck(hipCreateTextureObject(&texUv, &resDesc, &texDesc, NULL));

    Resize<YuvUnitx2> << <dim3((nDstWidth + 31) / 32, (nDstHeight + 31) / 32), dim3(16, 16) >> >(texY, texUv, dpDst, dpDstUV,
        nDstPitch, nDstWidth, nDstHeight, 1.0f * nDstWidth / nSrcWidth, 1.0f * nDstHeight / nSrcHeight);

    ck(hipDestroyTextureObject(texY));
    ck(hipDestroyTextureObject(texUv));
}

void ResizeNv12(unsigned char *dpDstNv12, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrcNv12, int nSrcPitch, int nSrcWidth, int nSrcHeight, unsigned char* dpDstNv12UV)
{
    unsigned char* dpDstUV = dpDstNv12UV ? dpDstNv12UV : dpDstNv12 + (nDstPitch*nDstHeight);
    return Resize<uchar2>(dpDstNv12, dpDstUV, nDstPitch, nDstWidth, nDstHeight, dpSrcNv12, nSrcPitch, nSrcWidth, nSrcHeight);
}


void ResizeP016(unsigned char *dpDstP016, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrcP016, int nSrcPitch, int nSrcWidth, int nSrcHeight, unsigned char* dpDstP016UV)
{
    unsigned char* dpDstUV = dpDstP016UV ? dpDstP016UV : dpDstP016 + (nDstPitch*nDstHeight);
    return Resize<ushort2>(dpDstP016, dpDstUV, nDstPitch, nDstWidth, nDstHeight, dpSrcP016, nSrcPitch, nSrcWidth, nSrcHeight);
}

static __global__ void Scale(hipTextureObject_t texSrc,
    uint8_t *pDst, int nPitch, int nWidth, int nHeight,
    float fxScale, float fyScale)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }

    *(unsigned char*)(pDst + (y * nPitch) + x) = (unsigned char)(fminf((tex2D<float>(texSrc, x * fxScale, y * fyScale)) * 255.0f, 255.0f));
}

static __global__ void Scale_uv(hipTextureObject_t texSrc,
    uint8_t *pDst, int nPitch, int nWidth, int nHeight,
    float fxScale, float fyScale)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x,
        y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= nWidth || y >= nHeight)
    {
        return;
    }

    float2 uv = tex2D<float2>(texSrc, x * fxScale, y * fyScale);
    uchar2 uvOut = uchar2{ (unsigned char)(fminf(uv.x * 255.0f, 255.0f)), (unsigned char)(fminf(uv.y * 255.0f, 255.0f)) };

    *(uchar2*)(pDst + (y * nPitch) + 2 * x) = uvOut;
}

void ScaleKernelLaunch(unsigned char *dpDst, int nDstPitch, int nDstWidth, int nDstHeight, unsigned char *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight, bool bUVPlane = false) 
{
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dpSrc;
    resDesc.res.pitch2D.desc = bUVPlane ? hipCreateChannelDesc<uchar2>() : hipCreateChannelDesc<unsigned char>();
    resDesc.res.pitch2D.width = nSrcWidth;
    resDesc.res.pitch2D.height = nSrcHeight;
    resDesc.res.pitch2D.pitchInBytes = nSrcPitch;

    hipTextureDesc texDesc = {};
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeNormalizedFloat;

    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;

    hipTextureObject_t texSrc = 0;
    ck(hipCreateTextureObject(&texSrc, &resDesc, &texDesc, NULL));

    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint32_t)nDstWidth + blockSize.x - 1) / blockSize.x, ((uint32_t)nDstHeight + blockSize.y - 1) / blockSize.y, 1);

    if (bUVPlane)
    {
        Scale_uv << <gridSize, blockSize >> >(texSrc, dpDst,
            nDstPitch, nDstWidth, nDstHeight, 1.0f * nSrcWidth / nDstWidth, 1.0f * nSrcHeight / nDstHeight);
    }
    else
    {
        Scale << <gridSize, blockSize >> >(texSrc, dpDst,
            nDstPitch, nDstWidth, nDstHeight, 1.0f * nSrcWidth / nDstWidth, 1.0f * nSrcHeight / nDstHeight);
    }

    ck(hipGetLastError());
    ck(hipDestroyTextureObject(texSrc));
}

void ScaleYUV420(unsigned char *dpDstY,
                 unsigned char* dpDstU,
                unsigned char* dpDstV,
                int nDstPitch,
                int nDstChromaPitch,
                int nDstWidth,
                int nDstHeight,
                unsigned char *dpSrcY,
                unsigned char* dpSrcU,
                unsigned char* dpSrcV, 
                int nSrcPitch,
                int nSrcChromaPitch,
                int nSrcWidth,
                int nSrcHeight,
                bool bSemiplanar)
{
    int chromaWidthDst = (nDstWidth + 1) / 2;
    int chromaHeightDst = (nDstHeight + 1) / 2;

    int chromaWidthSrc = (nSrcWidth + 1) / 2;
    int chromaHeightSrc = (nSrcHeight + 1) / 2;

    ScaleKernelLaunch(dpDstY, nDstPitch, nDstWidth, nDstHeight, dpSrcY, nSrcPitch, nSrcWidth, nSrcHeight);

    if (bSemiplanar)
    {
        ScaleKernelLaunch(dpDstU, nDstChromaPitch, chromaWidthDst, chromaHeightDst, dpSrcU, nSrcChromaPitch, chromaWidthSrc, chromaHeightSrc, true);
    }
    else
    {
        ScaleKernelLaunch(dpDstU, nDstChromaPitch, chromaWidthDst, chromaHeightDst, dpSrcU, nSrcChromaPitch, chromaWidthSrc, chromaHeightSrc);
        ScaleKernelLaunch(dpDstV, nDstChromaPitch, chromaWidthDst, chromaHeightDst, dpSrcV, nSrcChromaPitch, chromaWidthSrc, chromaHeightSrc);
    }
}
